#include "device.hpp"
#include "hip/hip_runtime.h"
#include "nvToolsExt.h"
#include "hip/hip_runtime_api.h"

DeviceMem::DeviceMem(std::size_t mem_size) : mMemSize(mem_size)
{
    checkCudaErrors(hipMalloc(static_cast<void**>(&mpDeviceBuf), mMemSize));
}

void* DeviceMem::GetDeviceBuffer() { return mpDeviceBuf; }

void DeviceMem::ToDevice(const void* p)
{
    checkCudaErrors(
        hipMemcpy(mpDeviceBuf, const_cast<void*>(p), mMemSize, hipMemcpyHostToDevice));
}

void DeviceMem::FromDevice(void* p)
{
    checkCudaErrors(hipMemcpy(p, mpDeviceBuf, mMemSize, hipMemcpyDeviceToHost));
}

DeviceMem::~DeviceMem() { checkCudaErrors(hipFree(mpDeviceBuf)); }

struct KernelTimerImpl
{
    KernelTimerImpl()
    {
        hipEventCreate(&mStart);
        hipEventCreate(&mEnd);
    }

    ~KernelTimerImpl()
    {
        hipEventDestroy(mStart);
        hipEventDestroy(mEnd);
    }

    void Start() { hipEventRecord(mStart, 0); }

    void End()
    {
        hipEventRecord(mEnd, 0);
        hipEventSynchronize(mEnd);
    }

    float GetElapsedTime() const
    {
        float time;
        hipEventElapsedTime(&time, mStart, mEnd);
        return time;
    }

    hipEvent_t mStart, mEnd;
};

KernelTimer::KernelTimer() : impl(new KernelTimerImpl()) {}

KernelTimer::~KernelTimer() {}

void KernelTimer::Start() { impl->Start(); }

void KernelTimer::End() { impl->End(); }

float KernelTimer::GetElapsedTime() const { return impl->GetElapsedTime(); }

void launch_kernel(const void* func, dim3 grid_dim, dim3 block_dim, void** args, float& time)
{
    KernelTimer timer;
    timer.Start();

    hipError_t error = hipLaunchKernel(reinterpret_cast<const void*>(func), grid_dim, block_dim, args, 0, 0);

    timer.End();
    time = timer.GetElapsedTime();

    checkCudaErrors(error);
}
