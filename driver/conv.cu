#include "hip/hip_runtime.h"
#include <iostream>
#include <numeric>
#include <initializer_list>
#include <cstdlib>
#include "nvToolsExt.h"
#include "tensor.hpp"
#include "constant_tensor_descriptor.cuh"
#include "device_direct_convolution_1.cuh"
#include "device_direct_convolution_2.cuh"

struct GeneratorConstant
{
    double value = 0;

    template <class... Is>
    double operator()(Is...)
    {
        return value;
    }
};

struct GeneratorTensor
{
    template <class... Is>
    double operator()(Is... is)
    {
#if 1
        return double(std::rand()) / double(RAND_MAX);
#elif 0
        return 1;
#elif 0
        std::initializer_list<std::size_t> ls = {static_cast<std::size_t>(is)...};
        return std::accumulate(ls.begin(), ls.end(), std::size_t(0));
#else
        assert(sizeof...(Is) > 0);
        std::initializer_list<std::size_t> ids = {static_cast<std::size_t>(is)...};
        std::vector<std::size_t> lens(sizeof...(Is), 100);
        std::vector<std::size_t> strides(sizeof...(Is), 1);
        std::partial_sum(lens.rbegin(), lens.rbegin() + (sizeof...(Is) - 1), strides.rbegin() + 1);
        return std::inner_product(ids.begin(), ids.end(), strides.begin(), std::size_t(0)) + 1;
#endif
    }
};

struct GeneratorTensor_2
{
    int min_value = 0;
    int max_value = 1;

    template <class... Is>
    double operator()(Is...)
    {
        return (std::rand() % (max_value - min_value)) + min_value;
    }
};

// this is ugly, only for 4d
template <class TConstTensorDesc>
void ostream_ConstantTensorDescriptor(TConstTensorDesc, std::ostream& os = std::cout)
{
    static_assert(TConstTensorDesc::nDim == 4, "nDim is not 4");

    constexpr auto I0   = Number<0>{};
    constexpr auto I1   = Number<1>{};
    constexpr auto I2   = Number<2>{};
    constexpr auto I3   = Number<3>{};
    constexpr auto desc = TConstTensorDesc{};

    os << "Lengths: {" << desc.GetLength(I0) << ", " << desc.GetLength(I1) << ", "
       << desc.GetLength(I2) << ", " << desc.GetLength(I3) << "}, "
       << "Strides: {" << desc.GetStride(I0) << ", " << desc.GetStride(I1) << ", "
       << desc.GetStride(I2) << ", " << desc.GetStride(I3) << "}" << std::endl;
}

// this is ugly, only for 4d
template <class TConstTensorDesc>
auto make_TensorDescriptor(TConstTensorDesc)
{
    static_assert(TConstTensorDesc::nDim == 4, "nDim is not 4");

    constexpr auto I0   = Number<0>{};
    constexpr auto I1   = Number<1>{};
    constexpr auto I2   = Number<2>{};
    constexpr auto I3   = Number<3>{};
    constexpr auto desc = TConstTensorDesc{};

    std::initializer_list<unsigned> lengths = {
        desc.GetLength(I0), desc.GetLength(I1), desc.GetLength(I2), desc.GetLength(I3)};
    std::initializer_list<unsigned> strides = {
        desc.GetStride(I0), desc.GetStride(I1), desc.GetStride(I2), desc.GetStride(I3)};

    return TensorDescriptor(lengths, strides);
}

template <class T>
void host_direct_convolution(const Tensor<T>& in, const Tensor<T>& wei, Tensor<T>& out)
{
    auto f = [&](auto n, auto k, auto ho, auto wo) {
        double v = 0;
        for(int c = 0; c < wei.mDesc.GetLengths()[1]; ++c)
        {
            for(int y = 0; y < wei.mDesc.GetLengths()[2]; ++y)
            {
                int hi = ho + y;
                for(int x = 0; x < wei.mDesc.GetLengths()[3]; ++x)
                {
                    int wi = wo + x;
                    v += in(n, c, hi, wi) * wei(k, c, y, x);
                }
            }
        }
        out(n, k, ho, wo) = v;
    };

    auto f_par = make_ParallelTensorFunctor(f,
                                            out.mDesc.GetLengths()[0],
                                            out.mDesc.GetLengths()[1],
                                            out.mDesc.GetLengths()[2],
                                            out.mDesc.GetLengths()[3]);

    f_par(std::thread::hardware_concurrency());
}

template <class T>
void host_winograd_3x3_convolution(const Tensor<T>& in, const Tensor<T>& wei, Tensor<T>& out)
{
    constexpr std::size_t OutTileSizeH = 2;
    constexpr std::size_t OutTileSizeW = 2;

    std::size_t N  = in.mDesc.GetLengths()[0];
    std::size_t C  = in.mDesc.GetLengths()[1];
    std::size_t HI = in.mDesc.GetLengths()[2];
    std::size_t WI = in.mDesc.GetLengths()[3];

    std::size_t K = wei.mDesc.GetLengths()[0];
    std::size_t S = wei.mDesc.GetLengths()[2];
    std::size_t R = wei.mDesc.GetLengths()[3];

    std::size_t HO = out.mDesc.GetLengths()[2];
    std::size_t WO = out.mDesc.GetLengths()[3];

    std::size_t InTileSizeH = OutTileSizeH + S - 1;
    std::size_t InTileSizeW = OutTileSizeW + R - 1;

    std::size_t Y = (HO + OutTileSizeH - 1) / OutTileSizeH;
    std::size_t X = (WO + OutTileSizeW - 1) / OutTileSizeW;

    Tensor<T> in_hold({N, C, Y, X, InTileSizeH, InTileSizeW});
    Tensor<T> in_transform({N, C, Y, X, InTileSizeH, InTileSizeW});
    Tensor<T> wei_transform({K, C, InTileSizeH, InTileSizeW});
    Tensor<T> out_transform({N, K, Y, X, InTileSizeH, InTileSizeH});
    Tensor<T> out_hold({N, K, Y, X, OutTileSizeH, OutTileSizeW});

    auto f_in_hold = [&](auto n, auto c, auto y, auto x) {
        for(int j = 0; j < InTileSizeH; ++j)
        {
            std::size_t hi = OutTileSizeH * y + j;
            for(int i = 0; i < InTileSizeW; ++i)
            {
                std::size_t wi            = OutTileSizeW * x + i;
                in_hold(n, c, y, x, j, i) = in(n, c, hi, wi);
            }
        }
    };

    auto f_in_transform = [&](auto n, auto c, auto y, auto x) {
        in_transform(n, c, y, x, 0, 0) = in_hold(n, c, y, x, 0, 0) - in_hold(n, c, y, x, 0, 2) -
                                         in_hold(n, c, y, x, 2, 0) + in_hold(n, c, y, x, 2, 2);
        in_transform(n, c, y, x, 0, 1) = in_hold(n, c, y, x, 0, 1) + in_hold(n, c, y, x, 0, 2) -
                                         in_hold(n, c, y, x, 2, 1) - in_hold(n, c, y, x, 2, 2);
        in_transform(n, c, y, x, 0, 2) = -in_hold(n, c, y, x, 0, 1) + in_hold(n, c, y, x, 0, 2) +
                                         in_hold(n, c, y, x, 2, 1) - in_hold(n, c, y, x, 2, 2);
        in_transform(n, c, y, x, 0, 3) = in_hold(n, c, y, x, 0, 1) - in_hold(n, c, y, x, 0, 3) -
                                         in_hold(n, c, y, x, 2, 1) + in_hold(n, c, y, x, 2, 3);

        in_transform(n, c, y, x, 1, 0) = in_hold(n, c, y, x, 1, 0) - in_hold(n, c, y, x, 1, 2) +
                                         in_hold(n, c, y, x, 2, 0) - in_hold(n, c, y, x, 2, 2);
        in_transform(n, c, y, x, 1, 1) = in_hold(n, c, y, x, 1, 1) + in_hold(n, c, y, x, 1, 2) +
                                         in_hold(n, c, y, x, 2, 1) + in_hold(n, c, y, x, 2, 2);
        in_transform(n, c, y, x, 1, 2) = -in_hold(n, c, y, x, 1, 1) + in_hold(n, c, y, x, 1, 2) -
                                         in_hold(n, c, y, x, 2, 1) + in_hold(n, c, y, x, 2, 2);
        in_transform(n, c, y, x, 1, 3) = in_hold(n, c, y, x, 1, 1) - in_hold(n, c, y, x, 1, 3) +
                                         in_hold(n, c, y, x, 2, 1) - in_hold(n, c, y, x, 2, 3);

        in_transform(n, c, y, x, 2, 0) = -in_hold(n, c, y, x, 1, 0) + in_hold(n, c, y, x, 1, 2) +
                                         in_hold(n, c, y, x, 2, 0) - in_hold(n, c, y, x, 2, 2);
        in_transform(n, c, y, x, 2, 1) = -in_hold(n, c, y, x, 1, 1) - in_hold(n, c, y, x, 1, 2) +
                                         in_hold(n, c, y, x, 2, 1) + in_hold(n, c, y, x, 2, 2);
        in_transform(n, c, y, x, 2, 2) = in_hold(n, c, y, x, 1, 1) - in_hold(n, c, y, x, 1, 2) -
                                         in_hold(n, c, y, x, 2, 1) + in_hold(n, c, y, x, 2, 2);
        in_transform(n, c, y, x, 2, 3) = -in_hold(n, c, y, x, 1, 1) + in_hold(n, c, y, x, 1, 3) +
                                         in_hold(n, c, y, x, 2, 1) - in_hold(n, c, y, x, 2, 3);

        in_transform(n, c, y, x, 3, 0) = in_hold(n, c, y, x, 1, 0) - in_hold(n, c, y, x, 1, 2) -
                                         in_hold(n, c, y, x, 3, 0) + in_hold(n, c, y, x, 3, 2);
        in_transform(n, c, y, x, 3, 1) = in_hold(n, c, y, x, 1, 1) + in_hold(n, c, y, x, 1, 2) -
                                         in_hold(n, c, y, x, 3, 1) - in_hold(n, c, y, x, 3, 2);
        in_transform(n, c, y, x, 3, 2) = -in_hold(n, c, y, x, 1, 1) + in_hold(n, c, y, x, 1, 2) +
                                         in_hold(n, c, y, x, 3, 1) - in_hold(n, c, y, x, 3, 2);
        in_transform(n, c, y, x, 3, 3) = in_hold(n, c, y, x, 1, 1) - in_hold(n, c, y, x, 1, 3) -
                                         in_hold(n, c, y, x, 3, 1) + in_hold(n, c, y, x, 3, 3);
    };

    auto f_wei_transform = [&](auto k, auto c) {
        wei_transform(k, c, 0, 0) = wei(k, c, 0, 0);
        wei_transform(k, c, 0, 1) =
            0.5 * wei(k, c, 0, 0) + 0.5 * wei(k, c, 0, 1) + 0.5 * wei(k, c, 0, 2);
        wei_transform(k, c, 0, 2) =
            0.5 * wei(k, c, 0, 0) - 0.5 * wei(k, c, 0, 1) + 0.5 * wei(k, c, 0, 2);
        wei_transform(k, c, 0, 3) = wei(k, c, 0, 2);

        wei_transform(k, c, 1, 0) =
            0.5 * wei(k, c, 0, 0) + 0.5 * wei(k, c, 1, 0) + 0.5 * wei(k, c, 2, 0);
        wei_transform(k, c, 1, 1) =
            0.25 * wei(k, c, 0, 0) + 0.25 * wei(k, c, 0, 1) + 0.25 * wei(k, c, 0, 2) +
            0.25 * wei(k, c, 1, 0) + 0.25 * wei(k, c, 1, 1) + 0.25 * wei(k, c, 1, 2) +
            0.25 * wei(k, c, 2, 0) + 0.25 * wei(k, c, 2, 1) + 0.25 * wei(k, c, 2, 2);
        wei_transform(k, c, 1, 2) =
            0.25 * wei(k, c, 0, 0) - 0.25 * wei(k, c, 0, 1) + 0.25 * wei(k, c, 0, 2) +
            0.25 * wei(k, c, 1, 0) - 0.25 * wei(k, c, 1, 1) + 0.25 * wei(k, c, 1, 2) +
            0.25 * wei(k, c, 2, 0) - 0.25 * wei(k, c, 2, 1) + 0.25 * wei(k, c, 2, 2);
        wei_transform(k, c, 1, 3) =
            0.5 * wei(k, c, 0, 2) + 0.5 * wei(k, c, 1, 2) + 0.5 * wei(k, c, 2, 2);

        wei_transform(k, c, 2, 0) =
            0.5 * wei(k, c, 0, 0) - 0.5 * wei(k, c, 1, 0) + 0.5 * wei(k, c, 2, 0);
        wei_transform(k, c, 2, 1) =
            0.25 * wei(k, c, 0, 0) + 0.25 * wei(k, c, 0, 1) + 0.25 * wei(k, c, 0, 2) -
            0.25 * wei(k, c, 1, 0) - 0.25 * wei(k, c, 1, 1) - 0.25 * wei(k, c, 1, 2) +
            0.25 * wei(k, c, 2, 0) + 0.25 * wei(k, c, 2, 1) + 0.25 * wei(k, c, 2, 2);
        wei_transform(k, c, 2, 2) =
            0.25 * wei(k, c, 0, 0) - 0.25 * wei(k, c, 0, 1) + 0.25 * wei(k, c, 0, 2) -
            0.25 * wei(k, c, 1, 0) + 0.25 * wei(k, c, 1, 1) - 0.25 * wei(k, c, 1, 2) +
            0.25 * wei(k, c, 2, 0) - 0.25 * wei(k, c, 2, 1) + 0.25 * wei(k, c, 2, 2);
        wei_transform(k, c, 2, 3) =
            0.5 * wei(k, c, 0, 2) - 0.5 * wei(k, c, 1, 2) + 0.5 * wei(k, c, 2, 2);

        wei_transform(k, c, 3, 0) = wei(k, c, 2, 0);
        wei_transform(k, c, 3, 1) =
            0.5 * wei(k, c, 2, 0) + 0.5 * wei(k, c, 2, 1) + 0.5 * wei(k, c, 2, 2);
        wei_transform(k, c, 3, 2) =
            0.5 * wei(k, c, 2, 0) - 0.5 * wei(k, c, 2, 1) + 0.5 * wei(k, c, 2, 2);
        wei_transform(k, c, 3, 3) = wei(k, c, 2, 2);
    };

    auto f_out_transform = [&](auto n, auto k, auto y, auto x) {
        for(int j = 0; j < InTileSizeH; ++j)
        {
            for(int i = 0; i < InTileSizeW; ++i)
            {
                double v = 0;
                for(int c = 0; c < C; ++c)
                {
                    v += in_transform(n, c, y, x, j, i) * wei_transform(k, c, j, i);
                }

                out_transform(n, k, y, x, j, i) = v;
            }
        }
    };

    auto f_out_hold = [&](auto n, auto k, auto y, auto x) {
        out_hold(n, k, y, x, 0, 0) =
            out_transform(n, k, y, x, 0, 0) + out_transform(n, k, y, x, 0, 1) +
            out_transform(n, k, y, x, 0, 2) + out_transform(n, k, y, x, 1, 0) +
            out_transform(n, k, y, x, 1, 1) + out_transform(n, k, y, x, 1, 2) +
            out_transform(n, k, y, x, 2, 0) + out_transform(n, k, y, x, 2, 1) +
            out_transform(n, k, y, x, 2, 2);
        out_hold(n, k, y, x, 0, 1) =
            out_transform(n, k, y, x, 0, 1) - out_transform(n, k, y, x, 0, 2) -
            out_transform(n, k, y, x, 0, 3) + out_transform(n, k, y, x, 1, 1) -
            out_transform(n, k, y, x, 1, 2) - out_transform(n, k, y, x, 1, 3) +
            out_transform(n, k, y, x, 2, 1) - out_transform(n, k, y, x, 2, 2) -
            out_transform(n, k, y, x, 2, 3);
        out_hold(n, k, y, x, 1, 0) =
            out_transform(n, k, y, x, 1, 0) + out_transform(n, k, y, x, 1, 1) +
            out_transform(n, k, y, x, 1, 2) - out_transform(n, k, y, x, 2, 0) -
            out_transform(n, k, y, x, 2, 1) - out_transform(n, k, y, x, 2, 2) -
            out_transform(n, k, y, x, 3, 0) - out_transform(n, k, y, x, 3, 1) -
            out_transform(n, k, y, x, 3, 2);
        out_hold(n, k, y, x, 1, 1) =
            out_transform(n, k, y, x, 1, 1) - out_transform(n, k, y, x, 1, 2) -
            out_transform(n, k, y, x, 1, 3) - out_transform(n, k, y, x, 2, 1) +
            out_transform(n, k, y, x, 2, 2) + out_transform(n, k, y, x, 2, 3) -
            out_transform(n, k, y, x, 3, 1) + out_transform(n, k, y, x, 3, 2) +
            out_transform(n, k, y, x, 3, 3);
    };

    auto f_out = [&](auto n, auto k, auto y, auto x) {
        for(int j = 0; j < OutTileSizeH; ++j)
        {
            std::size_t ho = OutTileSizeH * y + j;
            for(int i = 0; i < OutTileSizeW; ++i)
            {
                std::size_t wo    = OutTileSizeW * x + i;
                out(n, k, ho, wo) = out_hold(n, k, y, x, j, i);
            }
        }
    };

    std::size_t num_thread = std::thread::hardware_concurrency();

    make_ParallelTensorFunctor(f_in_hold, N, C, Y, X)(num_thread);
    make_ParallelTensorFunctor(f_in_transform, N, C, Y, X)(num_thread);
    make_ParallelTensorFunctor(f_wei_transform, K, C)(num_thread);
    make_ParallelTensorFunctor(f_out_transform, N, K, Y, X)(num_thread);
    make_ParallelTensorFunctor(f_out_hold, N, K, Y, X)(num_thread);
    make_ParallelTensorFunctor(f_out, N, K, Y, X)(num_thread);
}

template <class T>
void check_error(const Tensor<T>& ref, const Tensor<T>& result)
{
    float error     = 0;
    float max_diff  = 0;
    float ref_value = 0, result_value = 0;
    for(int i = 0; i < ref.mData.size(); ++i)
    {
        error += std::abs(ref.mData[i] - result.mData[i]);
        float diff = std::abs(ref.mData[i] - result.mData[i]);
        if(max_diff < diff)
        {
            max_diff     = diff;
            ref_value    = ref.mData[i];
            result_value = result.mData[i];
        }
    }

    std::cout << "error: " << error << std::endl;
    std::cout << "max_diff: " << max_diff << ", " << ref_value << ", " << result_value << std::endl;
}

int main()
{
#if 0
    constexpr unsigned N  = 1;
    constexpr unsigned C  = 1;
    constexpr unsigned HI = 34;
    constexpr unsigned WI = 34;
    constexpr unsigned K  = 1;
    constexpr unsigned S  = 3;
    constexpr unsigned R  = 3;
#elif 1
    constexpr unsigned N = 64;
    constexpr unsigned C = 256;
    constexpr unsigned HI = 34;
    constexpr unsigned WI = 34;
    constexpr unsigned K = 64;
    constexpr unsigned S = 3;
    constexpr unsigned R = 3;
#elif 0
    constexpr unsigned N = 1;
    constexpr unsigned C = 1;
    constexpr unsigned HI = 18;
    constexpr unsigned WI = 18;
    constexpr unsigned K = 1;
    constexpr unsigned S = 3;
    constexpr unsigned R = 3;
#elif 0
    constexpr unsigned N = 1;
    constexpr unsigned C = 1;
    constexpr unsigned HI = 4;
    constexpr unsigned WI = 4;
    constexpr unsigned K = 1;
    constexpr unsigned S = 3;
    constexpr unsigned R = 3;
#elif 0
    constexpr unsigned N  = 2;
    constexpr unsigned C  = 3;
    constexpr unsigned HI = 130;
    constexpr unsigned WI = 130;
    constexpr unsigned K  = 5;
    constexpr unsigned S  = 3;
    constexpr unsigned R  = 3;
#elif 0
    constexpr unsigned N  = 3;
    constexpr unsigned C  = 16;
    constexpr unsigned HI = 130;
    constexpr unsigned WI = 130;
    constexpr unsigned K  = 4;
    constexpr unsigned S  = 3;
    constexpr unsigned R  = 3;
#endif

    auto in_desc  = make_ConstantTensorDescriptor(Sequence<N, C, HI, WI>{});
    auto wei_desc = make_ConstantTensorDescriptor(Sequence<K, C, S, R>{});
    auto out_desc = get_output_4d_tensor_descriptor(in_desc, wei_desc);

    ostream_ConstantTensorDescriptor(in_desc, std::cout << "in_desc: ");
    ostream_ConstantTensorDescriptor(wei_desc, std::cout << "wei_desc: ");
    ostream_ConstantTensorDescriptor(out_desc, std::cout << "out_desc: ");

    Tensor<float> in(make_TensorDescriptor(in_desc));
    Tensor<float> wei(make_TensorDescriptor(wei_desc));
    Tensor<float> out_host(make_TensorDescriptor(out_desc));
    Tensor<float> out_device(make_TensorDescriptor(out_desc));

#if 1
    std::size_t num_thread = std::thread::hardware_concurrency();
    in.GenerateTensorValue(GeneratorTensor_2{-5, 5}, num_thread);
    wei.GenerateTensorValue(GeneratorTensor_2{-5, 5}, num_thread);
#endif

    for(int i = 0; i < 20; ++i)
    {
#if 1
        device_direct_convolution_1(in_desc, in, wei_desc, wei, out_desc, out_device);
#else
        device_winograd_convolution(in_desc, in, wei_desc, wei, out_desc, out_device);
#endif
    }

#if 1
    host_winograd_3x3_convolution(in, wei, out_host);
    check_error(out_host, out_device);
#elif 0
    host_direct_convolution(in, wei, out_host);
    check_error(out_host, out_device);
#endif

#if 0
    LogRange(std::cout << "in : ", in.mData, ",") << std::endl;
    LogRange(std::cout << "wei: ", wei.mData, ",") << std::endl;
    LogRange(std::cout << "out_host  : ", out_host.mData, ",") << std::endl;
    LogRange(std::cout << "out_device: ", out_device.mData, ",") << std::endl;
#endif
}
