#include "hip/hip_runtime.h"
#include <iostream>
#include <numeric>
#include <initializer_list>
#include <cstdlib>
#include "nvToolsExt.h"
#include "tensor.hpp"
#include "ConstantTensorDescriptor.cuh"
#include "conv_common.cuh"
#include "device_direct_convolution_1.cuh"
#include "device_direct_convolution_2.cuh"
#include "device_implicit_gemm_convolution_1_nchw_kcsr.cuh"
#include "device_implicit_gemm_convolution_1_nchw_srck_nkhw.cuh"
#include "device_implicit_gemm_convolution_1_chwn_csrk_khwn.cuh"
#include "device_implicit_gemm_convolution_1_chwn_csrk_khwn_padded.cuh"
#include "device_implicit_gemm_convolution_2_cnhw_srck_knhw.cuh"
#include "device_implicit_gemm_convolution_2_cnhw_csrk_knhw.cuh"
#include "device_implicit_gemm_convolution_2_cnhw_csrk_knhw_gemm_2.cuh"
//#include "device_winograd_convolution.cuh"

struct GeneratorTensor_1
{
    template <class... Is>
    double operator()(Is... is)
    {
        return 1;
    }
};

struct GeneratorTensor_2
{
    int min_value = 0;
    int max_value = 1;

    template <class... Is>
    double operator()(Is...)
    {
        return (std::rand() % (max_value - min_value)) + min_value;
    }
};

struct GeneratorTensor_3
{
    template <class... Is>
    double operator()(Is... is)
    {
#if 0
        std::initializer_list<std::size_t> ls = {static_cast<std::size_t>(is)...};
        return std::accumulate(ls.begin(), ls.end(), std::size_t(0));
#elif 1
        assert(sizeof...(Is) > 0);
        std::initializer_list<std::size_t> ids = {static_cast<std::size_t>(is)...};
        std::vector<std::size_t> lens(sizeof...(Is), 100);
        std::vector<std::size_t> strides(sizeof...(Is), 1);
        std::partial_sum(lens.rbegin(), lens.rbegin() + (sizeof...(Is) - 1), strides.rbegin() + 1);
        return std::inner_product(ids.begin(), ids.end(), strides.begin(), std::size_t(0)) + 1;
#endif
    }
};

struct GeneratorTensor_Checkboard
{
    template <class... Ts>
    double operator()(Ts... Xs) const
    {
        std::array<unsigned long, sizeof...(Ts)> dims = {{Xs...}};
        return std::accumulate(dims.begin(),
                               dims.end(),
                               true,
                               [](bool init, unsigned long x) -> int { return init != (x % 2); })
                   ? 1
                   : -1;
    }
};

// this is ugly, only for 4d
template <class TConstTensorDesc>
void ostream_ConstantTensorDescriptor(TConstTensorDesc, std::ostream& os = std::cout)
{
    static_assert(TConstTensorDesc::nDim == 4, "nDim is not 4");

    constexpr auto I0   = Number<0>{};
    constexpr auto I1   = Number<1>{};
    constexpr auto I2   = Number<2>{};
    constexpr auto I3   = Number<3>{};
    constexpr auto desc = TConstTensorDesc{};

    os << "Lengths: {" << desc.GetLength(I0) << ", " << desc.GetLength(I1) << ", "
       << desc.GetLength(I2) << ", " << desc.GetLength(I3) << "}, "
       << "Strides: {" << desc.GetStride(I0) << ", " << desc.GetStride(I1) << ", "
       << desc.GetStride(I2) << ", " << desc.GetStride(I3) << "}" << std::endl;
}

// this is ugly, only for 4d
template <class TConstTensorDesc>
auto make_TensorDescriptor(TConstTensorDesc)
{
    static_assert(TConstTensorDesc::nDim == 4, "nDim is not 4");

    constexpr auto I0   = Number<0>{};
    constexpr auto I1   = Number<1>{};
    constexpr auto I2   = Number<2>{};
    constexpr auto I3   = Number<3>{};
    constexpr auto desc = TConstTensorDesc{};

    std::initializer_list<unsigned> lengths = {
        desc.GetLength(I0), desc.GetLength(I1), desc.GetLength(I2), desc.GetLength(I3)};
    std::initializer_list<unsigned> strides = {
        desc.GetStride(I0), desc.GetStride(I1), desc.GetStride(I2), desc.GetStride(I3)};

    return TensorDescriptor(lengths, strides);
}

template <class T, class LowerPads, class UpperPads>
void host_direct_convolution(
    const Tensor<T>& in_nchw, const Tensor<T>& wei_kcsr, Tensor<T>& out, LowerPads, UpperPads)
{
    unsigned h_pad_low = LowerPads{}.Get(Number<0>{});
    unsigned w_pad_low = LowerPads{}.Get(Number<1>{});

    unsigned h_pad_up = UpperPads{}.Get(Number<0>{});
    unsigned w_pad_up = UpperPads{}.Get(Number<1>{});

    auto f = [&](auto n, auto k, auto ho, auto wo) {
        double v = 0;
        for(int c = 0; c < wei_kcsr.mDesc.GetLengths()[1]; ++c)
        {
            for(int y = 0; y < wei_kcsr.mDesc.GetLengths()[2]; ++y)
            {
                int hi = ho + y - h_pad_low;
                for(int x = 0; x < wei_kcsr.mDesc.GetLengths()[3]; ++x)
                {
                    int wi = wo + x - w_pad_low;
                    if(hi >= 0 && hi < in_nchw.mDesc.GetLengths()[2] && wi >= 0 &&
                       wi < in_nchw.mDesc.GetLengths()[3])
                    {
                        v += in_nchw(n, c, hi, wi) * wei_kcsr(k, c, y, x);
                    }
                }
            }
        }
        out(n, k, ho, wo) = v;
    };

    auto f_par = make_ParallelTensorFunctor(f,
                                            out.mDesc.GetLengths()[0],
                                            out.mDesc.GetLengths()[1],
                                            out.mDesc.GetLengths()[2],
                                            out.mDesc.GetLengths()[3]);

    f_par(std::thread::hardware_concurrency());
}

template <class T, class LowerPads, class UpperPads>
void host_winograd_3x3_convolution(
    const Tensor<T>& in_nchw, const Tensor<T>& wei_kcsr, Tensor<T>& out, LowerPads, UpperPads)
{
    constexpr std::size_t OutTileSizeH = 2;
    constexpr std::size_t OutTileSizeW = 2;

    std::size_t N  = in_nchw.mDesc.GetLengths()[0];
    std::size_t C  = in_nchw.mDesc.GetLengths()[1];
    std::size_t HI = in_nchw.mDesc.GetLengths()[2];
    std::size_t WI = in_nchw.mDesc.GetLengths()[3];

    std::size_t K = wei_kcsr.mDesc.GetLengths()[0];
    std::size_t S = wei_kcsr.mDesc.GetLengths()[2];
    std::size_t R = wei_kcsr.mDesc.GetLengths()[3];

    std::size_t HO = out.mDesc.GetLengths()[2];
    std::size_t WO = out.mDesc.GetLengths()[3];

    unsigned h_pad_low = LowerPads{}.Get(Number<0>{});
    unsigned w_pad_low = LowerPads{}.Get(Number<1>{});

    unsigned h_pad_up = UpperPads{}.Get(Number<0>{});
    unsigned w_pad_up = UpperPads{}.Get(Number<1>{});

    std::size_t InTileSizeH = OutTileSizeH + S - 1;
    std::size_t InTileSizeW = OutTileSizeW + R - 1;

    std::size_t Y = (HO + OutTileSizeH - 1) / OutTileSizeH;
    std::size_t X = (WO + OutTileSizeW - 1) / OutTileSizeW;

    Tensor<T> in_hold({N, C, Y, X, InTileSizeH, InTileSizeW});
    Tensor<T> in_transform({N, C, Y, X, InTileSizeH, InTileSizeW});
    Tensor<T> wei_transform({K, C, InTileSizeH, InTileSizeW});
    Tensor<T> out_transform({N, K, Y, X, InTileSizeH, InTileSizeH});
    Tensor<T> out_hold({N, K, Y, X, OutTileSizeH, OutTileSizeW});

    auto f_in_hold = [&](auto n, auto c, auto y, auto x) {
        for(int j = 0; j < InTileSizeH; ++j)
        {
            int hi = OutTileSizeH * y + j - h_pad_low;
            for(int i = 0; i < InTileSizeW; ++i)
            {
                int wi = OutTileSizeW * x + i - w_pad_low;

                if(hi >= 0 && hi < in_nchw.mDesc.GetLengths()[2] && wi >= 0 &&
                   wi < in_nchw.mDesc.GetLengths()[3])
                {
                    in_hold(n, c, y, x, j, i) = in_nchw(n, c, hi, wi);
                }
                else
                {
                    in_hold(n, c, y, x, j, i) = T(0);
                }
            }
        }
    };

    auto f_in_transform = [&](auto n, auto c, auto y, auto x) {
        in_transform(n, c, y, x, 0, 0) = in_hold(n, c, y, x, 0, 0) - in_hold(n, c, y, x, 0, 2) -
                                         in_hold(n, c, y, x, 2, 0) + in_hold(n, c, y, x, 2, 2);
        in_transform(n, c, y, x, 0, 1) = in_hold(n, c, y, x, 0, 1) + in_hold(n, c, y, x, 0, 2) -
                                         in_hold(n, c, y, x, 2, 1) - in_hold(n, c, y, x, 2, 2);
        in_transform(n, c, y, x, 0, 2) = -in_hold(n, c, y, x, 0, 1) + in_hold(n, c, y, x, 0, 2) +
                                         in_hold(n, c, y, x, 2, 1) - in_hold(n, c, y, x, 2, 2);
        in_transform(n, c, y, x, 0, 3) = in_hold(n, c, y, x, 0, 1) - in_hold(n, c, y, x, 0, 3) -
                                         in_hold(n, c, y, x, 2, 1) + in_hold(n, c, y, x, 2, 3);

        in_transform(n, c, y, x, 1, 0) = in_hold(n, c, y, x, 1, 0) - in_hold(n, c, y, x, 1, 2) +
                                         in_hold(n, c, y, x, 2, 0) - in_hold(n, c, y, x, 2, 2);
        in_transform(n, c, y, x, 1, 1) = in_hold(n, c, y, x, 1, 1) + in_hold(n, c, y, x, 1, 2) +
                                         in_hold(n, c, y, x, 2, 1) + in_hold(n, c, y, x, 2, 2);
        in_transform(n, c, y, x, 1, 2) = -in_hold(n, c, y, x, 1, 1) + in_hold(n, c, y, x, 1, 2) -
                                         in_hold(n, c, y, x, 2, 1) + in_hold(n, c, y, x, 2, 2);
        in_transform(n, c, y, x, 1, 3) = in_hold(n, c, y, x, 1, 1) - in_hold(n, c, y, x, 1, 3) +
                                         in_hold(n, c, y, x, 2, 1) - in_hold(n, c, y, x, 2, 3);

        in_transform(n, c, y, x, 2, 0) = -in_hold(n, c, y, x, 1, 0) + in_hold(n, c, y, x, 1, 2) +
                                         in_hold(n, c, y, x, 2, 0) - in_hold(n, c, y, x, 2, 2);
        in_transform(n, c, y, x, 2, 1) = -in_hold(n, c, y, x, 1, 1) - in_hold(n, c, y, x, 1, 2) +
                                         in_hold(n, c, y, x, 2, 1) + in_hold(n, c, y, x, 2, 2);
        in_transform(n, c, y, x, 2, 2) = in_hold(n, c, y, x, 1, 1) - in_hold(n, c, y, x, 1, 2) -
                                         in_hold(n, c, y, x, 2, 1) + in_hold(n, c, y, x, 2, 2);
        in_transform(n, c, y, x, 2, 3) = -in_hold(n, c, y, x, 1, 1) + in_hold(n, c, y, x, 1, 3) +
                                         in_hold(n, c, y, x, 2, 1) - in_hold(n, c, y, x, 2, 3);

        in_transform(n, c, y, x, 3, 0) = in_hold(n, c, y, x, 1, 0) - in_hold(n, c, y, x, 1, 2) -
                                         in_hold(n, c, y, x, 3, 0) + in_hold(n, c, y, x, 3, 2);
        in_transform(n, c, y, x, 3, 1) = in_hold(n, c, y, x, 1, 1) + in_hold(n, c, y, x, 1, 2) -
                                         in_hold(n, c, y, x, 3, 1) - in_hold(n, c, y, x, 3, 2);
        in_transform(n, c, y, x, 3, 2) = -in_hold(n, c, y, x, 1, 1) + in_hold(n, c, y, x, 1, 2) +
                                         in_hold(n, c, y, x, 3, 1) - in_hold(n, c, y, x, 3, 2);
        in_transform(n, c, y, x, 3, 3) = in_hold(n, c, y, x, 1, 1) - in_hold(n, c, y, x, 1, 3) -
                                         in_hold(n, c, y, x, 3, 1) + in_hold(n, c, y, x, 3, 3);
    };

    auto f_wei_transform = [&](auto k, auto c) {
        wei_transform(k, c, 0, 0) = wei_kcsr(k, c, 0, 0);
        wei_transform(k, c, 0, 1) =
            0.5 * wei_kcsr(k, c, 0, 0) + 0.5 * wei_kcsr(k, c, 0, 1) + 0.5 * wei_kcsr(k, c, 0, 2);
        wei_transform(k, c, 0, 2) =
            0.5 * wei_kcsr(k, c, 0, 0) - 0.5 * wei_kcsr(k, c, 0, 1) + 0.5 * wei_kcsr(k, c, 0, 2);
        wei_transform(k, c, 0, 3) = wei_kcsr(k, c, 0, 2);

        wei_transform(k, c, 1, 0) =
            0.5 * wei_kcsr(k, c, 0, 0) + 0.5 * wei_kcsr(k, c, 1, 0) + 0.5 * wei_kcsr(k, c, 2, 0);
        wei_transform(k, c, 1, 1) = 0.25 * wei_kcsr(k, c, 0, 0) + 0.25 * wei_kcsr(k, c, 0, 1) +
                                    0.25 * wei_kcsr(k, c, 0, 2) + 0.25 * wei_kcsr(k, c, 1, 0) +
                                    0.25 * wei_kcsr(k, c, 1, 1) + 0.25 * wei_kcsr(k, c, 1, 2) +
                                    0.25 * wei_kcsr(k, c, 2, 0) + 0.25 * wei_kcsr(k, c, 2, 1) +
                                    0.25 * wei_kcsr(k, c, 2, 2);
        wei_transform(k, c, 1, 2) = 0.25 * wei_kcsr(k, c, 0, 0) - 0.25 * wei_kcsr(k, c, 0, 1) +
                                    0.25 * wei_kcsr(k, c, 0, 2) + 0.25 * wei_kcsr(k, c, 1, 0) -
                                    0.25 * wei_kcsr(k, c, 1, 1) + 0.25 * wei_kcsr(k, c, 1, 2) +
                                    0.25 * wei_kcsr(k, c, 2, 0) - 0.25 * wei_kcsr(k, c, 2, 1) +
                                    0.25 * wei_kcsr(k, c, 2, 2);
        wei_transform(k, c, 1, 3) =
            0.5 * wei_kcsr(k, c, 0, 2) + 0.5 * wei_kcsr(k, c, 1, 2) + 0.5 * wei_kcsr(k, c, 2, 2);

        wei_transform(k, c, 2, 0) =
            0.5 * wei_kcsr(k, c, 0, 0) - 0.5 * wei_kcsr(k, c, 1, 0) + 0.5 * wei_kcsr(k, c, 2, 0);
        wei_transform(k, c, 2, 1) = 0.25 * wei_kcsr(k, c, 0, 0) + 0.25 * wei_kcsr(k, c, 0, 1) +
                                    0.25 * wei_kcsr(k, c, 0, 2) - 0.25 * wei_kcsr(k, c, 1, 0) -
                                    0.25 * wei_kcsr(k, c, 1, 1) - 0.25 * wei_kcsr(k, c, 1, 2) +
                                    0.25 * wei_kcsr(k, c, 2, 0) + 0.25 * wei_kcsr(k, c, 2, 1) +
                                    0.25 * wei_kcsr(k, c, 2, 2);
        wei_transform(k, c, 2, 2) = 0.25 * wei_kcsr(k, c, 0, 0) - 0.25 * wei_kcsr(k, c, 0, 1) +
                                    0.25 * wei_kcsr(k, c, 0, 2) - 0.25 * wei_kcsr(k, c, 1, 0) +
                                    0.25 * wei_kcsr(k, c, 1, 1) - 0.25 * wei_kcsr(k, c, 1, 2) +
                                    0.25 * wei_kcsr(k, c, 2, 0) - 0.25 * wei_kcsr(k, c, 2, 1) +
                                    0.25 * wei_kcsr(k, c, 2, 2);
        wei_transform(k, c, 2, 3) =
            0.5 * wei_kcsr(k, c, 0, 2) - 0.5 * wei_kcsr(k, c, 1, 2) + 0.5 * wei_kcsr(k, c, 2, 2);

        wei_transform(k, c, 3, 0) = wei_kcsr(k, c, 2, 0);
        wei_transform(k, c, 3, 1) =
            0.5 * wei_kcsr(k, c, 2, 0) + 0.5 * wei_kcsr(k, c, 2, 1) + 0.5 * wei_kcsr(k, c, 2, 2);
        wei_transform(k, c, 3, 2) =
            0.5 * wei_kcsr(k, c, 2, 0) - 0.5 * wei_kcsr(k, c, 2, 1) + 0.5 * wei_kcsr(k, c, 2, 2);
        wei_transform(k, c, 3, 3) = wei_kcsr(k, c, 2, 2);
    };

    auto f_out_transform = [&](auto n, auto k, auto y, auto x) {
        for(int j = 0; j < InTileSizeH; ++j)
        {
            for(int i = 0; i < InTileSizeW; ++i)
            {
                double v = 0;
                for(int c = 0; c < C; ++c)
                {
                    v += in_transform(n, c, y, x, j, i) * wei_transform(k, c, j, i);
                }

                out_transform(n, k, y, x, j, i) = v;
            }
        }
    };

    auto f_out_hold = [&](auto n, auto k, auto y, auto x) {
        out_hold(n, k, y, x, 0, 0) =
            out_transform(n, k, y, x, 0, 0) + out_transform(n, k, y, x, 0, 1) +
            out_transform(n, k, y, x, 0, 2) + out_transform(n, k, y, x, 1, 0) +
            out_transform(n, k, y, x, 1, 1) + out_transform(n, k, y, x, 1, 2) +
            out_transform(n, k, y, x, 2, 0) + out_transform(n, k, y, x, 2, 1) +
            out_transform(n, k, y, x, 2, 2);
        out_hold(n, k, y, x, 0, 1) =
            out_transform(n, k, y, x, 0, 1) - out_transform(n, k, y, x, 0, 2) -
            out_transform(n, k, y, x, 0, 3) + out_transform(n, k, y, x, 1, 1) -
            out_transform(n, k, y, x, 1, 2) - out_transform(n, k, y, x, 1, 3) +
            out_transform(n, k, y, x, 2, 1) - out_transform(n, k, y, x, 2, 2) -
            out_transform(n, k, y, x, 2, 3);
        out_hold(n, k, y, x, 1, 0) =
            out_transform(n, k, y, x, 1, 0) + out_transform(n, k, y, x, 1, 1) +
            out_transform(n, k, y, x, 1, 2) - out_transform(n, k, y, x, 2, 0) -
            out_transform(n, k, y, x, 2, 1) - out_transform(n, k, y, x, 2, 2) -
            out_transform(n, k, y, x, 3, 0) - out_transform(n, k, y, x, 3, 1) -
            out_transform(n, k, y, x, 3, 2);
        out_hold(n, k, y, x, 1, 1) =
            out_transform(n, k, y, x, 1, 1) - out_transform(n, k, y, x, 1, 2) -
            out_transform(n, k, y, x, 1, 3) - out_transform(n, k, y, x, 2, 1) +
            out_transform(n, k, y, x, 2, 2) + out_transform(n, k, y, x, 2, 3) -
            out_transform(n, k, y, x, 3, 1) + out_transform(n, k, y, x, 3, 2) +
            out_transform(n, k, y, x, 3, 3);
    };

    auto f_out = [&](auto n, auto k, auto y, auto x) {
        for(int j = 0; j < OutTileSizeH; ++j)
        {
            std::size_t ho = OutTileSizeH * y + j;
            for(int i = 0; i < OutTileSizeW; ++i)
            {
                std::size_t wo    = OutTileSizeW * x + i;
                out(n, k, ho, wo) = out_hold(n, k, y, x, j, i);
            }
        }
    };

    std::size_t num_thread = std::thread::hardware_concurrency();

    make_ParallelTensorFunctor(f_in_hold, N, C, Y, X)(num_thread);
    make_ParallelTensorFunctor(f_in_transform, N, C, Y, X)(num_thread);
    make_ParallelTensorFunctor(f_wei_transform, K, C)(num_thread);
    make_ParallelTensorFunctor(f_out_transform, N, K, Y, X)(num_thread);
    make_ParallelTensorFunctor(f_out_hold, N, K, Y, X)(num_thread);
    make_ParallelTensorFunctor(f_out, N, K, Y, X)(num_thread);
}

template <class T>
void check_error(const Tensor<T>& ref, const Tensor<T>& result)
{
    float error     = 0;
    float max_diff  = -1;
    float ref_value = 0, result_value = 0;
    for(int i = 0; i < ref.mData.size(); ++i)
    {
        error += std::abs(ref.mData[i] - result.mData[i]);
        float diff = std::abs(ref.mData[i] - result.mData[i]);
        if(max_diff < diff)
        {
            max_diff     = diff;
            ref_value    = ref.mData[i];
            result_value = result.mData[i];
        }
    }

    std::cout << "error: " << error << std::endl;
    std::cout << "max_diff: " << max_diff << ", " << ref_value << ", " << result_value << std::endl;
}

int main()
{
#if 0
    constexpr unsigned N  = 1;
    constexpr unsigned C  = 1;
    constexpr unsigned HI = 28;
    constexpr unsigned WI = 28;
    constexpr unsigned K  = 1;
    constexpr unsigned S  = 3;
    constexpr unsigned R  = 3;

    constexpr unsigned HPad = 0;
    constexpr unsigned WPad = 0;
#elif 0
    // 3x3, 34x34
    constexpr unsigned N = 64;
    constexpr unsigned C = 256;
    constexpr unsigned HI = 34;
    constexpr unsigned WI = 34;
    constexpr unsigned K = 64;
    constexpr unsigned S = 3;
    constexpr unsigned R = 3;

    constexpr unsigned HPad = 0;
    constexpr unsigned WPad = 0;
#elif 0
    // 3x3, 56x56
    constexpr unsigned N  = 64;
    constexpr unsigned C  = 64;
    constexpr unsigned HI = 56;
    constexpr unsigned WI = 56;
    constexpr unsigned K  = 64;
    constexpr unsigned S  = 3;
    constexpr unsigned R  = 3;
#elif 0
    // 3x3, 58x58
    constexpr unsigned N  = 64;
    constexpr unsigned C  = 64;
    constexpr unsigned HI = 58;
    constexpr unsigned WI = 58;
    constexpr unsigned K  = 64;
    constexpr unsigned S  = 3;
    constexpr unsigned R  = 3;
#elif 0
    // 5x5, 36x36
    constexpr unsigned N  = 64;
    constexpr unsigned C  = 256;
    constexpr unsigned HI = 36;
    constexpr unsigned WI = 36;
    constexpr unsigned K  = 64;
    constexpr unsigned S  = 5;
    constexpr unsigned R  = 5;
#elif 0
    // 7x7, 38x38
    constexpr unsigned N  = 64;
    constexpr unsigned C  = 256;
    constexpr unsigned HI = 38;
    constexpr unsigned WI = 38;
    constexpr unsigned K  = 64;
    constexpr unsigned S  = 7;
    constexpr unsigned R  = 7;
#elif 0
    // 3x3, 58x58
    constexpr unsigned N  = 16;
    constexpr unsigned C  = 128;
    constexpr unsigned HI = 58;
    constexpr unsigned WI = 58;
    constexpr unsigned K  = 256;
    constexpr unsigned S  = 3;
    constexpr unsigned R  = 3;
#elif 0
    // 3x3 filter, 58x58 image, 0x0 padding
    constexpr unsigned N  = 16;
    constexpr unsigned C  = 128;
    constexpr unsigned HI = 58;
    constexpr unsigned WI = 58;
    constexpr unsigned K  = 256;
    constexpr unsigned S  = 3;
    constexpr unsigned R  = 3;

    constexpr unsigned HPad = 0;
    constexpr unsigned WPad = 0;
#elif 0
    // 3x3 filter, 56x56 image, 1x1 padding
    constexpr unsigned N  = 16;
    constexpr unsigned C  = 128;
    constexpr unsigned HI = 56;
    constexpr unsigned WI = 56;
    constexpr unsigned K  = 256;
    constexpr unsigned S  = 3;
    constexpr unsigned R  = 3;

    constexpr unsigned HPad = 1;
    constexpr unsigned WPad = 1;
#elif 0
    // 3x3 filter, 28x28 image, 1x1 padding
    constexpr unsigned N  = 16;
    constexpr unsigned C  = 256;
    constexpr unsigned HI = 28;
    constexpr unsigned WI = 28;
    constexpr unsigned K  = 512;
    constexpr unsigned S  = 3;
    constexpr unsigned R  = 3;

    constexpr unsigned HPad = 1;
    constexpr unsigned WPad = 1;
#elif 1
    // 1x1 filter, 28x28 image
    constexpr unsigned N  = 16;
    constexpr unsigned C  = 256;
    constexpr unsigned HI = 28;
    constexpr unsigned WI = 28;
    constexpr unsigned K  = 512;
    constexpr unsigned S  = 1;
    constexpr unsigned R  = 1;

    constexpr unsigned HPad = 0;
    constexpr unsigned WPad = 0;
#elif 0
    // 3x3 filter, 20x84 image, 1x1 padding
    constexpr unsigned N  = 16;
    constexpr unsigned C  = 256;
    constexpr unsigned HI = 20;
    constexpr unsigned WI = 84;
    constexpr unsigned K  = 256;
    constexpr unsigned S  = 3;
    constexpr unsigned R  = 3;

    constexpr unsigned HPad = 1;
    constexpr unsigned WPad = 1;
#elif 0
    // 3x3 filter, 112x112 image, 1x1 padding
    constexpr unsigned N  = 16;
    constexpr unsigned C  = 64;
    constexpr unsigned HI = 112;
    constexpr unsigned WI = 112;
    constexpr unsigned K  = 128;
    constexpr unsigned S  = 3;
    constexpr unsigned R  = 3;

    constexpr unsigned HPad = 1;
    constexpr unsigned WPad = 1;
#elif 0
    // 5x5 filter, 20x86 image, 1x1 padding
    constexpr unsigned N  = 16;
    constexpr unsigned C  = 256;
    constexpr unsigned HI = 20;
    constexpr unsigned WI = 86;
    constexpr unsigned K  = 512;
    constexpr unsigned S  = 5;
    constexpr unsigned R  = 5;

    constexpr unsigned HPad = 1;
    constexpr unsigned WPad = 1;
#elif 0
    // 5x5 filter, 28x28 image, 2x2 padding
    constexpr unsigned N  = 16;
    constexpr unsigned C  = 192;
    constexpr unsigned HI = 28;
    constexpr unsigned WI = 28;
    constexpr unsigned K  = 32;
    constexpr unsigned S  = 5;
    constexpr unsigned R  = 5;

    constexpr unsigned HPad = 2;
    constexpr unsigned WPad = 2;
#endif

    auto lower_pads = Sequence<HPad, WPad>{};
    auto upper_pads = Sequence<HPad, WPad>{};

    auto in_nchw_desc  = make_ConstantTensorDescriptor(Sequence<N, C, HI, WI>{});
    auto wei_kcsr_desc = make_ConstantTensorDescriptor(Sequence<K, C, S, R>{});
    auto out_nkhw_desc = get_convolution_with_padding_output_default_4d_tensor_descriptor(
        in_nchw_desc, wei_kcsr_desc, lower_pads, upper_pads);

    ostream_ConstantTensorDescriptor(in_nchw_desc, std::cout << "in_nchw_desc: ");
    ostream_ConstantTensorDescriptor(wei_kcsr_desc, std::cout << "wei_kcsr_desc: ");
    ostream_ConstantTensorDescriptor(out_nkhw_desc, std::cout << "out_nkhw_desc: ");

    Tensor<float> in_nchw(make_TensorDescriptor(in_nchw_desc));
    Tensor<float> wei_kcsr(make_TensorDescriptor(wei_kcsr_desc));
    Tensor<float> out_nkhw_host(make_TensorDescriptor(out_nkhw_desc));
    Tensor<float> out_nkhw_device(make_TensorDescriptor(out_nkhw_desc));

    std::size_t num_thread = std::thread::hardware_concurrency();

#if 0
    in_nchw.GenerateTensorValue(GeneratorTensor_1{}, num_thread);
    wei_kcsr.GenerateTensorValue(GeneratorTensor_1{}, num_thread);
#elif 1
    in_nchw.GenerateTensorValue(GeneratorTensor_2{-5, 5}, num_thread);
    wei_kcsr.GenerateTensorValue(GeneratorTensor_2{-5, 5}, num_thread);
#elif 1
    in_nchw.GenerateTensorValue(GeneratorTensor_2{-2, 2}, num_thread);
    wei_kcsr.GenerateTensorValue(GeneratorTensor_1{}, num_thread);
#endif

    unsigned nrepeat = 100;

#if 1
#if 0
    device_direct_convolution_1
#elif 0
    device_direct_convolution_2
#elif 0
    device_implicit_gemm_convolution_1_nchw_kcsr
#elif 0
    device_implicit_gemm_convolution_1_nchw_srck_nkhw
#elif 0
    device_implicit_gemm_convolution_1_chwn_csrk_khwn
#elif 0
    device_implicit_gemm_convolution_2_cnhw_srck_knhw
#elif 0
    device_implicit_gemm_convolution_2_cnhw_csrk_knhw
#elif 1
    device_implicit_gemm_convolution_2_cnhw_csrk_knhw_gemm_2
#endif
    (in_nchw_desc, in_nchw, wei_kcsr_desc, wei_kcsr, out_nkhw_desc, out_nkhw_device, nrepeat);

#elif 1
    device_implicit_gemm_convolution_1_chwn_csrk_khwn_padded(in_nchw_desc,
                                                             in_nchw,
                                                             wei_kcsr_desc,
                                                             wei_kcsr,
                                                             out_nkhw_desc,
                                                             out_nkhw_device,
                                                             lower_pads,
                                                             upper_pads,
                                                             nrepeat);
#endif

#if 0
    if(S == 3 && R == 3)
    {
        host_winograd_3x3_convolution(in_nchw, wei_kcsr, out_nkhw_host, lower_pads, upper_pads);
    }
    else
    {
        host_direct_convolution(in_nchw, wei_kcsr, out_nkhw_host, lower_pads, upper_pads);
    }
    check_error(out_nkhw_host, out_nkhw_device);
#endif

#if 0
    LogRange(std::cout << "in_nchw : ", in_nchw.mData, ",") << std::endl;
    LogRange(std::cout << "wei_kcsr: ", wei_kcsr.mData, ",") << std::endl;
    LogRange(std::cout << "out_nkhw_host  : ", out_nkhw_host.mData, ",") << std::endl;
    LogRange(std::cout << "out_nkhw_device: ", out_nkhw_device.mData, ",") << std::endl;
#endif
}
